#include "hip/hip_runtime.h"
/*
 * (c) 2019 Gustavo Valiente gustavo.valiente@protonmail.com
 *
 * MIT License, see LICENSE file.
 */

#include "pcps_thrust_normal_extractor.h"

#include "thrust/transform.h"
#include "thrust/device_vector.h"
#include "thrust/iterator/counting_iterator.h"
#include "pcps_thrust_cached_allocator.h"

namespace pcps_thrust
{

// suppress "function was declared but never referenced warning"
// https://stackoverflow.com/questions/49836419/how-to-hide-nvccs-function-was-declared-but-never-referenced-warnings
#pragma push
#pragma diag_suppress 177

namespace
{
    using DeviceVector = thrust::device_vector<float4>;


    struct NormalMeanTransform
    {
        DeviceVector::const_iterator normalsBegin;
        int normalRegionX;
        int normalRegionY;
        int normalRegionWidth;
        int normalsWidth;

        __device__
        float4 operator()(int index) const noexcept
        {
            // C = A % B is equivalent to C = A – B * (A / B)
            int aDivB = index / normalRegionWidth;
            // int aModB = index % normalRegionWidth;
            int aModB = index - (aDivB * normalRegionWidth);
            int indexX = normalRegionX + aModB;
            int indexY = normalRegionY + aDivB;
            int globalIndex = (indexY * normalsWidth) + indexX;
            return *(normalsBegin + globalIndex);
        }
    };


    struct NormalMeanReduce
    {
        __device__
        float4 operator()(float4 a, float4 b) const noexcept
        {
            return { a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w };
        }
    };


    struct NormalStdDevTransform
    {
        float4 mean;
        DeviceVector::const_iterator normalsBegin;
        int normalRegionX;
        int normalRegionY;
        int normalRegionWidth;
        int normalsWidth;

        __device__
        float operator()(int index) const noexcept
        {
            // C = A % B is equivalent to C = A – B * (A / B)
            int aDivB = index / normalRegionWidth;
            // int aModB = index % normalRegionWidth;
            int aModB = index - (aDivB * normalRegionWidth);
            int indexX = normalRegionX + aModB;
            int indexY = normalRegionY + aDivB;
            int globalIndex = (indexY * normalsWidth) + indexX;
            float4 normal = *(normalsBegin + globalIndex);
            float stdDev;

            if(normal.w > 0)
            {
                float dotProduct = normal.x * mean.x + normal.y * mean.y + normal.z * mean.z;
                stdDev = std::acos(dotProduct);
            }
            else
            {
                stdDev = 0;
            }

            return stdDev;
        }
    };
}

#pragma pop

void getNormalMean(const void* normals, int normalRegionX, int normalRegionY, int normalRegionWidth,
                   int normalRegionHeight, int normalsWidth, float4& mean, int& numValidNormals,
                   CachedAllocator& allocator)
{
    auto par = thrust::cuda::par(allocator);
    auto normalVector = static_cast<const DeviceVector*>(normals);
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + (normalRegionWidth * normalRegionHeight);
    NormalMeanTransform transform{ normalVector->begin(), normalRegionX, normalRegionY, normalRegionWidth,
                normalsWidth };
    float4 sum = thrust::transform_reduce(par, first, last, transform, float4{ 0, 0, 0, 0 }, NormalMeanReduce());
    numValidNormals = int(std::round(sum.w));

    if(numValidNormals)
    {
        mean = { sum.x / numValidNormals, sum.y / numValidNormals, sum.z / numValidNormals, 1 };
    }
    else
    {
        mean = { 0, 0, 0, 0 };
    }
}

void getNormalStdDev(const void* normals, int normalRegionX, int normalRegionY, int normalRegionWidth,
                     int normalRegionHeight, int normalsWidth, float4 mean, int numValidNormals, float& stdDev,
                     CachedAllocator& allocator)
{
    auto par = thrust::cuda::par(allocator);
    auto normalVector = static_cast<const DeviceVector*>(normals);
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + (normalRegionWidth * normalRegionHeight);
    NormalStdDevTransform transform{ mean, normalVector->begin(), normalRegionX, normalRegionY, normalRegionWidth,
                normalsWidth };
    float sum = thrust::transform_reduce(par, first, last, transform, 0.0f, thrust::plus<float>());
    stdDev = sum / numValidNormals;
}

}
