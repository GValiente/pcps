#include "hip/hip_runtime.h"
/*
 * (c) 2019 Gustavo Valiente gustavo.valiente@protonmail.com
 *
 * MIT License, see LICENSE file.
 */

#include "pcps_thrust_organizer.h"

#include <cmath>
#include "thrust/transform.h"
#include "thrust/device_vector.h"
#include "thrust/transform_reduce.h"
#include "thrust/iterator/counting_iterator.h"
#include "pcps_thrust_cached_allocator.h"

namespace pcps_thrust
{

// suppress "function was declared but never referenced warning"
// https://stackoverflow.com/questions/49836419/how-to-hide-nvccs-function-was-declared-but-never-referenced-warnings
#pragma push
#pragma diag_suppress 177

namespace
{
    using PointIterator = thrust::device_vector<float2>::const_iterator;

    struct MinMaxTransform
    {
        __device__
        float4 operator()(float2 point) const noexcept
        {
            return { point.x, point.y, point.x, point.y };
        }
    };

    struct MinMaxReduce
    {
        __device__
        float4 operator()(float4 a, float4 b) const noexcept
        {
            float minX = a.x < b.x ? a.x : b.x;
            float minY = a.y < b.y ? a.y : b.y;
            float maxX = a.z > b.z ? a.z : b.z;
            float maxY = a.w > b.w ? a.w : b.w;
            return { minX, minY, maxX, maxY };
        }
    };

    struct DistanceTransform
    {
        PointIterator pointsBegin;
        int numPoints;
        float maxDistance;

        __device__
        float operator()(int index) const noexcept
        {
            PointIterator pointsIt = pointsBegin + index;
            float2 point = *pointsIt;
            float minDistance = maxDistance;
            ++index;
            ++pointsIt;

            while(index < numPoints)
            {
                float2 otherPoint = *pointsIt;
                float distanceX = point.x - otherPoint.x;
                float distanceY = point.y - otherPoint.y;
                float distance = (distanceX * distanceX) + (distanceY * distanceY);

                if(distance < minDistance)
                {
                    minDistance = distance;
                }

                ++index;
                ++pointsIt;
            }

            return minDistance;
        }
    };

    struct DistanceReduce
    {
        __device__
        float operator()(float a, float b) const noexcept
        {
            return a < b ? a : b;
        }
    };

    struct IndexTransform
    {
        float minX;
        float minY;
        int width;
        float cellSizeInv;

        __device__
        int operator()(float2 point) const noexcept
        {
            float relativeX = point.x - minX;
            float relativeY = point.y - minY;
            auto col = int(relativeX * cellSizeInv);
            auto row = int(relativeY * cellSizeInv);
            return (row * width) + col;
        }
    };
}

#pragma pop

void organize(const thrust::host_vector<float2>& points2D, int maximumWidth, int maximumHeight, float epsilon,
              int& width, int& height, thrust::host_vector<int>& indices, CachedAllocator& allocator)
{
    auto par = thrust::cuda::par(allocator);
    thrust::device_vector<float2> devicePoints2D = points2D;
    std::size_t numPoints = points2D.size();

    // Retrieve bounding box:
    auto maxValue = std::numeric_limits<float>::max();
    float4 initialBoundingBox = { maxValue, maxValue, -maxValue, -maxValue };
    float4 boundingBox = thrust::transform_reduce(par, devicePoints2D.begin(), devicePoints2D.end(), MinMaxTransform(),
                                                  initialBoundingBox, MinMaxReduce());

    // Retrieve minimum distance between points:

    float minX = boundingBox.x;
    float minY = boundingBox.y;
    float distanceX = boundingBox.z - minX;
    float distanceY = boundingBox.w - minY;
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + long(numPoints);
    float minDistance = thrust::transform_reduce(par, first, last,
                                                 DistanceTransform{ devicePoints2D.begin(), int(numPoints), maxValue },
                                                 maxValue, DistanceReduce());
    minDistance = std::sqrt(minDistance);

    // Retrieve output cloud size and cell size:

    if(minDistance > epsilon)
    {
        width = std::min(int(distanceX / minDistance) + 1, maximumWidth);
        height = std::min(int(distanceY / minDistance) + 1, maximumHeight);
    }
    else
    {
        width = maximumWidth;
        height = maximumHeight;
    }

    float cellSize = std::max(distanceX / width, distanceY / height);
    float cellSizeInv = 1 / cellSize;
    width = int(distanceX * cellSizeInv) + 1;
    height = int(distanceY * cellSizeInv) + 1;

    // Fill output cloud points vector:

    thrust::device_vector<int> deviceIndices(numPoints);
    thrust::transform(par, devicePoints2D.begin(), devicePoints2D.end(), deviceIndices.begin(),
                      IndexTransform{ minX, minY, width, cellSizeInv});
    indices.clear();
    indices.reserve(numPoints);
    thrust::copy(deviceIndices.begin(), deviceIndices.end(), indices.begin());
}

}
