#include "hip/hip_runtime.h"
/*
 * (c) 2019 Gustavo Valiente gustavo.valiente@protonmail.com
 *
 * MIT License, see LICENSE file.
 */

#include "pcps_thrust_cloud.h"

#include "thrust/device_vector.h"

namespace pcps_thrust
{

void* createDevicePoints(const float4* hostPoints, std::size_t numHostPoints)
{
    auto devicePoints = new thrust::device_vector<float4>(numHostPoints);
    thrust::copy(hostPoints, hostPoints + numHostPoints, devicePoints->begin());
    return devicePoints;
}

void updateHostPoints(const void* devicePoints, float4* hostPoints)
{
    auto deviceVector = static_cast<const thrust::device_vector<float4>*>(devicePoints);
    thrust::copy(deviceVector->begin(), deviceVector->end(), hostPoints);
}

void destroyDevicePoints(void* devicePoints) noexcept
{
    auto deviceVector = static_cast<thrust::device_vector<float4>*>(devicePoints);
    delete deviceVector;
}

}
