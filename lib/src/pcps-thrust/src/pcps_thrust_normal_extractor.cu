#include "hip/hip_runtime.h"
/*
 * (c) 2019 Gustavo Valiente gustavo.valiente@protonmail.com
 *
 * MIT License, see LICENSE file.
 */

#include "pcps_thrust_normal_extractor.h"

#include <cmath>
#include "thrust/transform.h"
#include "thrust/device_vector.h"
#include "thrust/iterator/counting_iterator.h"
#include "pcps_thrust_cached_allocator.h"

namespace pcps_thrust
{

// suppress "function was declared but never referenced warning"
// https://stackoverflow.com/questions/49836419/how-to-hide-nvccs-function-was-declared-but-never-referenced-warnings
#pragma push
#pragma diag_suppress 177

namespace
{
    using DeviceVector = thrust::device_vector<float4>;

    __device__
    float3 cross(float3 a, float3 b)
    {
        return float3{ a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x };
    }

    __device__
    float3 divide(float3 a, float divisor)
    {
        return float3{ a.x / divisor, a.y / divisor, a.z / divisor };
    }

    struct Transform
    {
        float4 flipViewPoint;
        DeviceVector::const_iterator pointsBegin;
        int neighborLevels;
        int cols;
        int rows;
        float epsilon;

        __device__
        float4 operator()(int index) const noexcept
        {
            // computeCovarianceMatrix:

            int pointCount = 0;
            float accum[9] = { 0 };
            int x0 = index % cols;
            int y0 = index / cols;
            int r2 = neighborLevels * neighborLevels;
            int xMin = max(-neighborLevels, -x0);
            int xMax = min(neighborLevels, cols - x0 - 1);
            int yMin = max(-neighborLevels, -y0);
            int yMax = min(neighborLevels, rows - y0 - 1);

            for(int y = yMin; y <= yMax; ++y)
            {
                int pointIndex = ((y0 + y) * cols) + (x0 + xMin);
                int y2 = y * y;

                for(int x = xMin; x <= xMax; ++x)
                {
                    if((x * x) + y2 <= r2)
                    {
                        auto pointIt = pointsBegin + pointIndex;
                        float4 point = *pointIt;
                        float pX = point.x;
                        float pY = point.y;
                        float pZ = point.z;

                        if(! isnan(pX) && ! isnan(pY) && ! isnan(pZ))
                        {
                            accum[0] += pX * pX;
                            accum[1] += pX * pY;
                            accum[2] += pX * pZ;
                            accum[3] += pY * pY;
                            accum[4] += pY * pZ;
                            accum[5] += pZ * pZ;
                            accum[6] += pX;
                            accum[7] += pY;
                            accum[8] += pZ;
                            ++pointCount;
                        }
                    }

                    ++pointIndex;
                }
            }

            if(! pointCount)
            {
                return float4{ 0, 0, 0, 0 };
            }

            for(int index = 0; index < 9; ++index)
            {
                accum[index] /= pointCount;
            }

            float covarianceMatrix[9];
            covarianceMatrix[0] = accum[0] - accum[6] * accum[6];
            covarianceMatrix[1] = accum[1] - accum[6] * accum[7];
            covarianceMatrix[2] = accum[2] - accum[6] * accum[8];
            covarianceMatrix[4] = accum[3] - accum[7] * accum[7];
            covarianceMatrix[5] = accum[4] - accum[7] * accum[8];
            covarianceMatrix[8] = accum[5] - accum[8] * accum[8];
            covarianceMatrix[3] = covarianceMatrix[1];
            covarianceMatrix[6] = covarianceMatrix[2];
            covarianceMatrix[7] = covarianceMatrix[5];

            // Scale the matrix so its entries are in [-1,1].
            // The scaling is applied only when at least one matrix entry has magnitude larger than 1:

            float scale = fabs(covarianceMatrix[0]);

            for(int index = 1; index < 9; ++index)
            {
                scale = max(fabs(covarianceMatrix[index]), scale);
            }

            if(scale > epsilon)
            {
                float scaleInv = 1 / scale;

                for(int index = 0; index < 9; ++index)
                {
                    covarianceMatrix[index] *= scaleInv;
                }
            }

            // computeFirstRoot:

            float m00 = covarianceMatrix[0];
            float m01 = covarianceMatrix[1];
            float m02 = covarianceMatrix[2];
            float m11 = covarianceMatrix[4];
            float m12 = covarianceMatrix[5];
            float m22 = covarianceMatrix[8];

            // The characteristic equation is x^3 - c2*x^2 + c1*x - c0 = 0.
            // The eigenvalues are the roots to this equation, all guaranteed to be real-valued,
            // because the matrix is symmetric:

            float c0 = m00 * m11 * m22 +
                    2.0f * m01 * m02 * m12 -
                    m00 * m12 * m12 -
                    m11 * m02 * m02 -
                    m22 * m01 * m01;

            float eigenFirstValue = 0;

            if(fabs(c0) > epsilon)
            {
                float c1 = m00 * m11 -
                        m01 * m01 +
                        m00 * m22 -
                        m02 * m02 +
                        m11 * m22 -
                        m12 * m12;

                float c2 = m00 + m11 + m22;

                const float s_inv3 = 1.0f / 3.0f;
                const float s_sqrt3 = 1.732050808f;

                // Construct the parameters used in classifying the roots of the equation
                // and in solving the equation for the roots in closed form:

                float c2_over_3 = c2 * s_inv3;
                float half_b = 0.5f * (c0 + c2_over_3 * (2.0f * c2_over_3 * c2_over_3 - c1));
                float q = half_b * half_b;
                float a_over_3 = (c1 - c2 * c2_over_3) * s_inv3;
                float rho;
                float q_sqrt;

                if(a_over_3 > 0)
                {
                    a_over_3 = 0;
                    rho = 0;
                }
                else
                {
                    rho = sqrt(-a_over_3);
                    q += a_over_3 * a_over_3 * a_over_3;
                }

                if(q > 0)
                {
                    q_sqrt = 0.0f;
                }
                else
                {
                    q_sqrt = sqrt(-q);
                }

                // Compute the eigenvalues by solving for the roots of the polynomial:

                float theta = atan2(q_sqrt, half_b) * s_inv3;
                float cos_theta = cos(theta);
                float sin_theta = sin(theta);
                float roots0 = c2_over_3 + 2.0f * rho * cos_theta;
                float roots1 = c2_over_3 - rho * (cos_theta + s_sqrt3 * sin_theta);
                float roots2 = c2_over_3 - rho * (cos_theta - s_sqrt3 * sin_theta);

                // Sort in increasing order:

                float temp;

                if(roots0 >= roots1)
                {
                    temp = roots0;
                    roots0 = roots1;
                    roots1 = temp;
                }

                if(roots1 >= roots2)
                {
                    temp = roots1;
                    roots1 = roots2;
                    roots2 = temp;

                    if(roots0 >= roots1)
                    {
                        temp = roots0;
                        roots0 = roots1;
                        roots1 = temp;
                    }
                }

                // eigenval for symetric positive semi-definite matrix can not be negative! Set it to 0:

                if(roots0 > 0)
                {
                    eigenFirstValue = roots0;
                }
            }

            covarianceMatrix[0] -= eigenFirstValue;
            covarianceMatrix[4] -= eigenFirstValue;
            covarianceMatrix[8] -= eigenFirstValue;

            // eigen33:

            float3 row0{ covarianceMatrix[0], covarianceMatrix[1], covarianceMatrix[2] };
            float3 row1{ covarianceMatrix[3], covarianceMatrix[4], covarianceMatrix[5] };
            float3 row2{ covarianceMatrix[6], covarianceMatrix[7], covarianceMatrix[8] };

            float3 vec1 = cross(row0, row1);
            float3 vec2 = cross(row0, row2);
            float3 vec3 = cross(row1, row2);

            float len1 = (vec1.x * vec1.x) + (vec1.y * vec1.y) + (vec1.z * vec1.z);
            float len2 = (vec2.x * vec2.x) + (vec2.y * vec2.y) + (vec2.z * vec2.z);
            float len3 = (vec3.x * vec3.x) + (vec3.y * vec3.y) + (vec3.z * vec3.z);

            float3 normal;

            if(len1 >= len2 && len1 >= len3)
            {
                normal = vec1;
                normal = divide(normal, sqrt(len1));
            }
            else if(len2 >= len1 && len2 >= len3)
            {
                normal = vec2;
                normal = divide(normal, sqrt(len2));
            }
            else
            {
                normal = vec3;
                normal = divide(normal, sqrt(len3));
            }

            if(std::abs(normal.x) > epsilon || std::abs(normal.y) > epsilon || std::abs(normal.z) > epsilon)
            {
                float4 result{ normal.x, normal.y, normal.z, 1 };

                // Flip the estimated normal of a point towards the given viewpoint:

                float4 point = *(pointsBegin + index);
                float4 viewPoint = flipViewPoint;
                viewPoint.x -= point.x;
                viewPoint.y -= point.y;
                viewPoint.z -= point.z;

                float cosTheta = (viewPoint.x * normal.x + viewPoint.y * normal.y + viewPoint.z * normal.z);

                if(cosTheta < 0)
                {
                    result.x *= -1;
                    result.y *= -1;
                    result.z *= -1;
                }

                return result;
            }
            else
            {
                return float4{ 0, 0, 0, 0 };
            }
        }
    };
}

#pragma pop

void computeNormals(const void* inputPoints, float4 flipViewPoint, int cols, int rows, int neighborLevels,
                    float epsilon, void* outputNormals, CachedAllocator& allocator)
{
    auto par = thrust::cuda::par(allocator);
    auto inputPointVector = static_cast<const DeviceVector*>(inputPoints);
    auto outputNormalVector = static_cast<DeviceVector*>(outputNormals);
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last = first + (cols * rows);
    thrust::transform(par, first, last, outputNormalVector->begin(),
                      Transform{ flipViewPoint, inputPointVector->begin(), neighborLevels, cols, rows, epsilon });
}

}
